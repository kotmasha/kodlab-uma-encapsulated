#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "Agent.h"
#include "worker.h"

int worker::t=0;
hipEvent_t start,stop;
//helper function
/*
*/
__host__ __device__ int compi_GPU(int x){
	if(x%2==0) return x+1;
	else return x-1;
}

__host__ __device__ int ind(int row,int col,int width){
	return row*width+col;
}

__global__ void conjunction_kernel(bool *b1,bool *b2,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		b1[index]=b1[index]&&b2[index];
	}
}

__global__ void disjunction_kernel(bool *b1,bool *b2,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		b1[index]=b1[index]||b2[index];
	}
}


__global__ void negate_disjunction_star_kernel(bool *b1,bool *b2,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		if(index%2==0){
			b1[index]=b1[index]&&!b2[index+1];
		}
		else{
			b1[index]=b1[index]&&!b2[index-1];
		}
	}
}

__global__ void int2bool_kernel(bool *b,int *i,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		if(i[index]==1) b[index]=true;
		else b[index]=false;
	}
}

__global__ void bool2int_kernel(int *i,bool *b,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		if(b[index]) i[index]=1;
		else i[index]=0;
	}
}

__host__ __device__ bool *worker_dir(worker &worker,bool compiY,bool compiX,bool isSymmetry){
	if(!isSymmetry){
		if(!compiY&&!compiX) return worker.dij;
		else if(!compiY&&compiX) return worker.di_j;
		else if(compiY&&!compiX) return worker.d_ij;
		else return worker.d_i_j;
	}
	else{
		if(!compiX&&!compiY) return worker.dji;
		else if(!compiX&&compiY) return worker.dj_i;
		else if(compiX&&!compiY) return worker.d_ji;
		else return worker.d_j_i;
	}
}

__host__ __device__ double *worker_weight(worker &worker,bool compiY,bool compiX){
	if(!compiY&&!compiX) return worker.wij;
	else if(!compiY&&compiX) return worker.wi_j;
	else if(compiY&&!compiX) return worker.w_ij;
	else return worker.w_i_j;
}

//helper function
__device__ bool implies_GPU(worker &worker,bool y,bool x){//implies
	double rc=*(worker_weight(worker,y,x));
	double r_c=*(worker_weight(worker,!y,x));
	double rc_=*(worker_weight(worker,y,!x));
	double r_c_=*(worker_weight(worker,!y,!x));
	double epsilon=(rc+r_c+rc_+r_c_)*worker.threshold;
	double m=min(epsilon,min(rc,min(r_c,r_c_)));
	return rc_<m;
}

__device__ bool equivalent_GPU(worker &worker,bool y,bool x){//equivalent
	double rc=*(worker_weight(worker,y,x));
	double r_c=*(worker_weight(worker,!y,x));
	double rc_=*(worker_weight(worker,y,!x));
	double r_c_=*(worker_weight(worker,!y,!x));
	double epsilon=(rc+r_c+rc_+r_c_)*worker.threshold;
	return rc_==0&&r_c==0;
}

__device__ void orient_square_GPU(worker &worker){//orient_square
	if(worker.sensor_id1==worker.sensor_id2) return;
	*(worker_dir(worker,false,false,false))=false;
	*(worker_dir(worker,false,true,false))=false;
	*(worker_dir(worker,true,false,false))=false;
	*(worker_dir(worker,true,true,false))=false;
	*(worker_dir(worker,false,false,true))=false;
	*(worker_dir(worker,false,true,true))=false;
	*(worker_dir(worker,true,false,true))=false;
	*(worker_dir(worker,true,true,true))=false;

	int square_is_oriented=0;
	for(int x=0;x<2;++x){
		for(int y=0;y<2;++y){
			if(square_is_oriented==0){
				if(implies_GPU(worker,y,x)){
					*(worker_dir(worker,y,x,false))=true;
					*(worker_dir(worker,!y,!x,true))=true;
					*(worker_dir(worker,y,x,true))=false;
					*(worker_dir(worker,!y,!x,false))=false;
					*(worker_dir(worker,!y,x,true))=false;
					*(worker_dir(worker,!y,x,false))=false;
					*(worker_dir(worker,y,!x,false))=false;
					*(worker_dir(worker,y,!x,true))=false;
                    square_is_oriented=1;
				}//implies
				if(equivalent_GPU(worker,y,x)){
					*(worker_dir(worker,y,x,false))=true;
					*(worker_dir(worker,y,x,true))=true;
					*(worker_dir(worker,!y,!x,true))=true;
					*(worker_dir(worker,!y,!x,false))=true;
					*(worker_dir(worker,!y,x,true))=false;
					*(worker_dir(worker,!y,x,false))=false;
					*(worker_dir(worker,y,!x,false))=false;
					*(worker_dir(worker,y,!x,true))=false;
                    square_is_oriented=1;
				}//equivalent
			}//square_is_oriented
		}//j
	}//i
}

__global__ void update_weights_kernel_empirical(worker *workers,bool *observe,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		int y=workers[index].sensor_id1;
		int x=workers[index].sensor_id2;

		*(workers[index].wij)+=observe[2*y]*observe[2*x];
		*(workers[index].w_ij)+=observe[2*y+1]*observe[2*x];
		*(workers[index].wi_j)+=observe[2*y]*observe[2*x+1];
		*(workers[index].w_i_j)+=observe[2*y+1]*observe[2*x+1];
	}
}

__global__ void update_weights_kernel_discounted(worker *workers,bool *observe,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		int y=workers[index].sensor_id1;
		int x=workers[index].sensor_id2;
		double q=workers[index].q;
		*(workers[index].wij)=*(workers[index].wij)*q+(1-q)*observe[2*y]*observe[2*x];
		*(workers[index].w_ij)=*(workers[index].w_ij)*q+(1-q)*observe[2*y+1]*observe[2*x];
		*(workers[index].wi_j)=*(workers[index].wi_j)*q+(1-q)*observe[2*y]*observe[2*x+1];
		*(workers[index].w_i_j)=*(workers[index].w_i_j)*q+(1-q)*observe[2*y+1]*observe[2*x+1];
	}
}

__global__ void calculate_sensor_value(worker *workers,float *sensor_value,int size){//gather all sensor value,workerSize
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		int y=workers[index].sensor_id1;
		int x=workers[index].sensor_id2;
		atomicAdd(sensor_value+2*y,*(workers[index].wij)+*(workers[index].wi_j));
		atomicAdd(sensor_value+2*y+1,*(workers[index].w_ij)+*(workers[index].w_i_j));
		atomicAdd(sensor_value+2*x,*(workers[index].wij)+*(workers[index].w_ij));
		atomicAdd(sensor_value+2*x+1,*(workers[index].wi_j)+*(workers[index].w_i_j));
	}
}

__global__ void update_weights_kernel_distributed(worker *workers,float *sensor_value,bool *observe,int size,int sensorSize,int t){//workerSize
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		int y=workers[index].sensor_id1;
		int x=workers[index].sensor_id2;
		double tij=(*(workers[index].wij)*t+observe[2*y]*observe[2*x])/(t+1);
		double t_ij=(*(workers[index].w_ij)*t+observe[2*y+1]*observe[2*x])/(t+1);
		double ti_j=(*(workers[index].wi_j)*t+observe[2*y]*observe[2*x+1])/(t+1);
		double t_i_j=(*(workers[index].w_i_j)*t+observe[2*y+1]*observe[2*x+1])/(t+1);
		double sij=-*(workers[index].wij)+(sensor_value[2*y]+sensor_value[2*x])/2;
		double s_ij=-*(workers[index].w_ij)+(sensor_value[2*y+1]+sensor_value[2*x])/2;
		double si_j=-*(workers[index].wi_j)+(sensor_value[2*y]+sensor_value[2*x+1])/2;
		double s_i_j=-*(workers[index].w_i_j)+(sensor_value[2*y+1]+sensor_value[2*x+1])/2;

		*(workers[index].wij)=(tij+sij)/(2*sensorSize-3);
		*(workers[index].w_ij)=(t_ij+s_ij)/(2*sensorSize-3);
		*(workers[index].wi_j)=(ti_j+si_j)/(2*sensorSize-3);
		*(workers[index].w_i_j)=(t_i_j+s_i_j)/(2*sensorSize-3);

		//*(workers[index].wij)=tij;
		//*(workers[index].w_ij)=t_ij;
		//*(workers[index].wi_j)=ti_j;
		//*(workers[index].w_i_j)=t_i_j;
	}
}

__global__ void orient_all_kernel(worker *workers,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		orient_square_GPU(workers[index]);
	}
}

__global__ void multiply_kernel(worker *worker,bool *data,bool *affected_worker,int size,int measurableSize){
	int index=threadIdx.x;
	extern __shared__ bool shared[];
	bool *xs=&shared[0];
	bool *ys=&shared[measurableSize];
	__shared__ bool flag[1];
	int x,y;
	int j=index;
	while(j<size){
		x=worker[j].sensor_id2;
		y=worker[j].sensor_id1;
		xs[2*x]=data[2*x];xs[2*x+1]=data[2*x+1];
		xs[2*y]=data[2*y];xs[2*y+1]=data[2*y+1];
		ys[2*x]=data[2*x];ys[2*x+1]=data[2*x+1];
		ys[2*y]=data[2*y];ys[2*y+1]=data[2*y+1];
		j+=512;
	}
	flag[0]=true;
	__syncthreads();
	while(flag[0]){
		flag[0]=false;
		j=index;
		__syncthreads();
		while(j<size){
			x=worker[j].sensor_id2;
			y=worker[j].sensor_id1;
			if(ys[2*x]==0&&xs[2*y]==1&&(*worker[j].dij)) ys[2*x]=1;
			if(ys[2*x+1]==0&&xs[2*y]==1&&(*worker[j].di_j)) ys[2*x+1]=1;
			if(ys[2*x]==0&&xs[2*y+1]==1&&(*worker[j].d_ij)) ys[2*x]=1;
			if(ys[2*x+1]==0&&xs[2*y+1]==1&&(*worker[j].d_i_j)) ys[2*x+1]=1;
			if(ys[2*y]==0&&xs[2*x]==1&&(*worker[j].dji)) ys[2*y]=1;
			if(ys[2*y+1]==0&&xs[2*x]==1&&(*worker[j].dj_i)) ys[2*y+1]=1;
			if(ys[2*y]==0&&xs[2*x+1]==1&&(*worker[j].d_ji)) ys[2*y]=1;
			if(ys[2*y+1]==0&&xs[2*x+1]==1&&(*worker[j].d_j_i)) ys[2*y+1]=1;
			j+=512;
		}
		j=index;
		__syncthreads();
		while(j<size){
			x=worker[j].sensor_id2;
			y=worker[j].sensor_id1;
			if(ys[2*y]==1&&xs[2*y]==0) flag[0]=true;
			if(ys[2*y+1]==1&&xs[2*y+1]==0) flag[0]=true;
			if(ys[2*x]==1&&xs[2*x]==0) flag[0]=true;
			if(ys[2*x+1]==1&&xs[2*x+1]==0) flag[0]=true;
			xs[2*y]=ys[2*y];
			xs[2*y+1]=ys[2*y+1];
			xs[2*x]=ys[2*x];
			xs[2*x+1]=ys[2*x+1];
			j+=512;
		}
		__syncthreads();
	}
	j=index;
	__syncthreads();
	while(j<size){
		x=worker[j].sensor_id2;
		y=worker[j].sensor_id1;
		data[2*x]=ys[2*x];data[2*x+1]=ys[2*x+1];
		data[2*y]=ys[2*y];data[2*y+1]=ys[2*y+1];
		if(affected_worker!=NULL&&ys[2*x]||ys[2*x+1]||ys[2*y]||ys[2*y+1]) affected_worker[j]=true;
		j+=512;
	}
}

//mask=Signal([(ind in actions_list) for ind in xrange(self._SIZE)])
__global__ void mask_kernel(bool *mask,int *actionlist,int size){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<size){
		for(int i=0;i<size;++i){
			if(index==actionlist[i]){
				mask[index]=true;
				return;
			}
		}
		mask[index]=false;
	}
}

//before invoke this function make sure dev_load and dev_signal have correct data
//the computed data will be in dev_load
void Agent::propagate_GPU(){//propagate
	hipMemset(dev_affected_worker,0,workerSize*sizeof(bool));

	multiply_kernel<<<1, 512, 2*measurableSize*sizeof(bool)>>>(dev_worker,dev_load,dev_affected_worker,workerSize,measurableSize);

	multiply_kernel<<<1, 512, 2*measurableSize*sizeof(bool)>>>(dev_worker,dev_signal,dev_affected_worker,workerSize,measurableSize);

	// standard operations
	disjunction_kernel<<<(measurableSize+255)/256,256>>>(dev_load,dev_signal,measurableSize);
	negate_disjunction_star_kernel<<<(measurableSize+255)/256,256>>>(dev_load,dev_signal,measurableSize);
	
	hipMemcpy(Gload,dev_load,measurableSize*sizeof(bool),hipMemcpyDeviceToHost);
	hipMemcpy(Gaffected_worker,dev_affected_worker,workerSize*sizeof(bool),hipMemcpyDeviceToHost);
}

void Agent::setSignal(vector<bool> observe){//this is where data comes in in every frame
	for(int i=0;i<observe.size();++i){
		Gobserve[i]=observe[i];
	}
	hipMemcpy(dev_observe,Gobserve,measurableSize*sizeof(bool),hipMemcpyHostToDevice);
}

void Agent::update_weights(){}

void Agent_Empirical::update_weights(){
	update_weights_kernel_empirical<<<(workerSize+255)/256,256>>>(dev_worker,dev_observe,workerSize);
}

void Agent_Distributed::update_weights(){
	hipMemset(dev_sensor_value,0.0,measurableSize*sizeof(float));
	calculate_sensor_value<<<(workerSize+255)/256,256>>>(dev_worker,dev_sensor_value,workerSize);
	update_weights_kernel_distributed<<<(workerSize+255)/256,256>>>(dev_worker,dev_sensor_value,dev_observe,workerSize,sensorSize,worker::t);
}

void Agent_Discounted::update_weights(){
	update_weights_kernel_discounted<<<(workerSize+255)/256,256>>>(dev_worker,dev_observe,workerSize);
}

void Agent::update_state_GPU(bool mode){//true for decide
	float dt=0;
	is_log_on=true;
	if(is_log_on){
		//hipEventCreate(&start);
		hipEventRecord(start);
	}
	update_weights();
	n_update_weight++;
	if(is_log_on){
		//hipEventCreate(&stop);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&dt, start,stop);
		t_update_weight+=dt;
	}
	worker::add_time();//when distributed or multiply agents, need to move it to upper loop
	//update_weight
	
	if(mode){
		if(is_log_on){
			//hipEventCreate(&start);
			hipEventRecord(start);
		}
		orient_all_kernel<<<(workerSize+255)/256,256>>>(dev_worker,workerSize);
		n_orient_all++;
		if(is_log_on){
			//hipEventCreate(&stop);
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&dt, start,stop);
			t_orient_all+=dt;
		}
	}//orient_all

	hipMemcpy(dev_signal,dev_observe,measurableSize*sizeof(bool),hipMemcpyDeviceToDevice);
	hipMemset(dev_load,false,measurableSize*sizeof(bool));
	if(is_log_on){
		//hipEventCreate(&start);
		hipEventRecord(start);
	}
	propagate_GPU();
	n_propagation++;
	if(is_log_on){
		//hipEventCreate(&stop);
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&dt, start,stop);
		t_propagation+=dt;
	}
	hipMemcpy(Gcurrent,dev_load,measurableSize*sizeof(bool),hipMemcpyDeviceToHost);
	hipMemcpy(dev_current,dev_load,measurableSize*sizeof(bool),hipMemcpyDeviceToDevice);
	hipMemcpy(Gdir,dev_dir,measurableSize*measurableSize*sizeof(bool),hipMemcpyDeviceToHost);
}

void Agent::halucinate_GPU(vector<int> actions_list){
	//mask=Signal([(ind in actions_list) for ind in xrange(self._SIZE)])
	vector<bool> mask=initMask(actions_list);
	vector<int> v;
	for(int i=0;i<actions_list.size();++i){
		for(int j=0;j<measurableSize;++j){
			if(context.find(pair<int,int>(actions_list[i],j))!=context.end()&&Gcurrent[j]){
				v.push_back(context[pair<int,int>(actions_list[i],j)]);
			}
		}
	}
	//relevant_pairs=[(act,ind) for act in actions_list for ind in xrange(self._SIZE) if (act,ind) in self._CONTEXT and self._CURRENT.value(ind)]
	//map(mask.set,[self._CONTEXT[i,j] for i,j in relevant_pairs],[True for i,j in relevant_pairs])
	for(int i=0;i<v.size();++i) mask[v[i]]=true;
	
	for(int i=0;i<mask.size();++i){
		Gmask[i]=mask[i];
	}
	hipMemcpy(dev_mask,Gmask,measurableSize*sizeof(bool),hipMemcpyHostToDevice);
	//copy data
	hipMemcpy(dev_signal,dev_mask,measurableSize*sizeof(bool),hipMemcpyDeviceToDevice);
	hipMemcpy(dev_load,dev_current,measurableSize*sizeof(bool),hipMemcpyDeviceToDevice);
	propagate_GPU();
	//return self.propagate(mask,self._CURRENT)
}

void Agent::freeData(){//free data in case of memory leak
	delete[] Gdir;
	delete[] Gweights;
	delete[] Gthresholds;
	delete[] Gobserve;
	delete[] Gdfs;
	delete[] Gsignal;
	delete[] Gload;

	delete[] Gmask;
	delete[] Gcurrent;
	delete[] Gworker;
	delete[] Gaffected_worker;

	hipFree(dev_worker);
	hipFree(dev_dir);
	hipFree(dev_thresholds);
	hipFree(dev_weights);
	hipFree(dev_observe);
	hipFree(dev_dfs);
	hipFree(dev_signal);
	hipFree(dev_load);
	
	hipFree(out_load);
	hipFree(out_signal);

	hipFree(dev_scan);

	hipFree(dev_mask);
	hipFree(dev_current);

	hipFree(dev_sensor_value);

	hipFree(dev_affected_worker);
}

void Agent::initData(string name,int sensorSize,vector<vector<int> > context_key,vector<int> context_value,
		vector<string> sensors_names,vector<string> evals_names,vector<vector<int> > generalized_actions){
	//data init
	this->name=name;
	this->sensorSize=sensorSize;
	this->measurableSize=2*sensorSize;
	this->workerSize=sensorSize*(sensorSize-1)/2;
	this->sensors_names=sensors_names;
	this->evals_names=evals_names;
	this->generalized_actions=generalized_actions;
	srand (time(NULL));
	for(int i=0;i<measurableSize;++i){
		name_to_num[sensors_names[i]]=i;
	}
	if(Gdir!=NULL){
		freeData();
	}
	
	Gdir=new bool[measurableSize*measurableSize];
	Gweights=new double[measurableSize*measurableSize];
	Gthresholds=new double[measurableSize*measurableSize];
	Gobserve=new bool[measurableSize];
	Gdfs=new bool[1];
	Gsignal=new bool[measurableSize];
	Gload=new bool[measurableSize];

	Gmask=new bool[measurableSize];
	Gcurrent=new bool[measurableSize];
	Gworker=new worker[workerSize];
	Gaffected_worker=new bool[workerSize];
	
	hipMalloc(&dev_dir,measurableSize*measurableSize*sizeof(bool));
	hipMalloc(&dev_thresholds,measurableSize*measurableSize*sizeof(double));
	hipMalloc(&dev_weights,measurableSize*measurableSize*sizeof(double));
	hipMalloc(&dev_observe,measurableSize*sizeof(bool));
	hipMalloc(&dev_dfs,sizeof(bool));
	hipMalloc(&dev_signal,measurableSize*sizeof(bool));
	hipMalloc(&dev_load,measurableSize*sizeof(bool));
	hipMalloc(&dev_sensor_value,measurableSize*sizeof(float));

	initWorkerMemory(dev_weights,dev_dir);
	hipMalloc(&dev_worker,workerSize*sizeof(worker));
	hipMemcpy(dev_worker,Gworker,workerSize*sizeof(worker),hipMemcpyHostToDevice);

	hipMalloc(&out_signal, measurableSize*sizeof(int));
	hipMalloc(&out_load, measurableSize*sizeof(int));

	hipMalloc(&dev_mask,measurableSize*sizeof(bool));
	hipMalloc(&dev_current,measurableSize*sizeof(bool));

	hipMalloc(&dev_scan,measurableSize*sizeof(int));

	hipMalloc(&dev_affected_worker,workerSize*sizeof(bool));

	// we need to make diagonals of Gdir = 1, true
	for(int i=0;i<measurableSize;++i){
		for(int j=0;j<measurableSize;++j){
			Gthresholds[i*measurableSize+j]=threshold;
			Gweights[i*measurableSize+j]=0.0;
			Gdir[i*measurableSize+j]=false;
			// new implementation
			if(i == j)
				Gdir[i*measurableSize+j] = true;
		}
	}

	hipMemcpy(dev_thresholds,Gthresholds,measurableSize*measurableSize*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_weights,Gweights,measurableSize*measurableSize*sizeof(double),hipMemcpyHostToDevice);
	hipMemcpy(dev_dir,Gdir,measurableSize*measurableSize*sizeof(bool),hipMemcpyHostToDevice);
	//init threshold

	for(int i=0;i<context_key.size();++i){
		context[pair<int,int>(context_key[i][0],context_key[i][1])]=context_value[i];
	}

	hipEventCreate(&start);
	hipEventCreate(&stop);
	cout<<"succeed"<<endl;
}

void Agent::initWorkerMemory(double *weights,bool *dir){
	int y=0,x=y+1;
	for(int i=0;i<workerSize;++i){
		Gworker[i]=worker("","",y,x);//name input required
		Gworker[i].wij=&weights[ind(2*y,2*x,measurableSize)];
		Gworker[i].w_ij=&weights[ind(2*y+1,2*x,measurableSize)];
		Gworker[i].wi_j=&weights[ind(2*y,2*x+1,measurableSize)];
		Gworker[i].w_i_j=&weights[ind(2*y+1,2*x+1,measurableSize)];

		Gworker[i].dij=&dir[ind(2*y,2*x,measurableSize)];
		Gworker[i].d_ij=&dir[ind(2*y+1,2*x,measurableSize)];
		Gworker[i].di_j=&dir[ind(2*y,2*x+1,measurableSize)];
		Gworker[i].d_i_j=&dir[ind(2*y+1,2*x+1,measurableSize)];

		Gworker[i].dji=&dir[ind(2*x,2*y,measurableSize)];
		Gworker[i].d_ji=&dir[ind(2*x+1,2*y,measurableSize)];
		Gworker[i].dj_i=&dir[ind(2*x,2*y+1,measurableSize)];
		Gworker[i].d_j_i=&dir[ind(2*x+1,2*y+1,measurableSize)];

		x++;
		if(x==sensorSize){
			y++;
			x=y+1;
		}
		Gworker[i].threshold=threshold;
	}
}

void Agent_Empirical::initWorkerMemory(double *weights,bool *dir){
	Agent::initWorkerMemory(weights,dir);
}

void Agent_Distributed::initWorkerMemory(double *weights,bool *dir){
	Agent::initWorkerMemory(weights,dir);
}

void Agent_Discounted::initWorkerMemory(double *weights,bool *dir){
	Agent::initWorkerMemory(weights,dir);
	for(int i=0;i<workerSize;++i){
		Gworker[i].q=q;
	}
}