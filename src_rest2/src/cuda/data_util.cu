#include "data_util.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

void data_util::boolD2D(bool *from, bool *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(bool), hipMemcpyDeviceToDevice);
}

void data_util::boolH2D(bool *from, bool *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(bool), hipMemcpyHostToDevice);
}

void data_util::boolD2H(bool *from, bool *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(bool), hipMemcpyDeviceToHost);
}

void data_util::intD2D(int *from, int *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(int), hipMemcpyDeviceToDevice);
}

void data_util::intH2D(int *from, int *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(int), hipMemcpyHostToDevice);
}

void data_util::intD2H(int *from, int *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(int), hipMemcpyDeviceToHost);
}

void data_util::doubleD2D(double *from, double *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(double), hipMemcpyDeviceToDevice);
}

void data_util::doubleH2D(double *from, double *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(double), hipMemcpyHostToDevice);
}

void data_util::doubleD2H(double *from, double *to, int size, int from_offset, int to_offset) {
	hipMemcpy(to + to_offset, from + from_offset, size * sizeof(double), hipMemcpyDeviceToHost);
}