#include "hip/hip_runtime.h"
#include "kernel.h"
#include "kernel_util.cuh"
#include "device_util.h"
#include "Logger.h"

static Logger kernelUtilLogger("KernelUtil", "log/device.log");

/*
---------------------HELPER FUNCTION-----------------------
*/

/*
set the input vector to be all true
Input: bool array, and size of it
*/
__global__ void alltrue_kernel(bool *b, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b[index] = true;
	}
}

/*
set the input array to be all false
Input: bool array, and size of it
*/
__global__ void allfalse_kernel(bool *b, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b[index] = false;
	}
}

/*
convert the input bool to int, false->0, true->1
Input: a bool source array, and int dest array, and size of both array
*/
__global__ void bool2int_kernel(bool *b, int *i, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		if (b[index]) i[index] = 1;
		else i[index] = 0;
	}
}

/*
convert the input bool to double, false->0, true->1
Input: a bool source array, and double dest array, and size of both array
*/
__global__ void bool2double_kernel(bool *b, double *d, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		if (b[index]) d[index] = 1.0;
		else d[index] = 0.0;
	}
}

/*
This function does the conjunction for two lists
Input: two bool lists, and size of both
Output: None
*/
__global__ void conjunction_kernel(bool *b1, bool *b2, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b1[index] = b1[index] && b2[index];
	}
}

/*
This function computes the difference of two lists
Input: two bool lists (same length)
Output: None
*/
__global__ void subtraction_kernel(bool *b1, bool *b2, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b1[index] = b1[index] && !b2[index];
	}
}

/*
This function does the disjunction for two lists
Input: two bool lists
Output: None
*/
__global__ void disjunction_kernel(bool *b1, bool *b2, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b1[index] = b1[index] || b2[index];
	}
}

/*
This function does compi, conjunction together
Input: two boolean lists
Output: None
*/
__global__ void negateConjunctionStar_kernel(bool *b1, bool *b2, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b1[index] = b1[index] && !b2[compi(index)];
	}
}

/*
Conjuncate the start of array, store the result in first array
Input: two bool array, and size of both
*/
__global__ void conjunctionStar_kernel(bool *b1, bool *b2, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b1[index] = b1[index] && b2[compi(index)];
	}
}

/*
Changing the signal from up to down
Input: two bool array, and both size
*/
__global__ void up2down_kernel(bool *b1, bool *b2, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		b2[index] = !b1[compi(index)];
	}
}

__global__ void sum_kernel(double *d, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int offset = 1;
	int j = index;
	while (true) {
		j = index;
		while (j < size) {
			if (j % (2 * offset) == 0 && j + offset < size) d[j] += d[j + offset];
			j += THREAD1D;
		}
		offset = offset * 2;
		__syncthreads();
		if (offset >= size) break;
	}
}

__global__ void initMaskSignal_kernel(bool *b, int initSize, int size) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	if (index < size) {
		if (index < 2 * initSize) {
			b[index] = true;
		}
		else {
			b[index] = false;
		}
	}
}

void kernel_util::alltrue(bool *b, int size) {
	cudaCheckErrors("check alltrue error");
	alltrue_kernel << <GRID1D(size), BLOCK1D >> > (b, size);
	kernelUtilLogger.debug("alltrue_kernel invoked");
	cudaCheckErrors("check alltrue error");
}

void kernel_util::allfalse(bool *b, int size) {
	allfalse_kernel << <GRID1D(size), BLOCK1D >> > (b, size);
	kernelUtilLogger.debug("allfalse_kernel invoked");
	cudaCheckErrors("check allfalse error");
}

void kernel_util::bool2int(bool *b, int *i, int size) {
	bool2int_kernel << <GRID1D(size), BLOCK1D >> > (b, i, size);
	kernelUtilLogger.debug("bool2int_kernel invoked");
	cudaCheckErrors("check bool2int error");
}

void kernel_util::bool2double(bool *b, double *d, int size) {
	bool2double_kernel << <GRID1D(size), BLOCK1D >> > (b, d, size);
	kernelUtilLogger.debug("bool2double_kernel invoked");
	cudaCheckErrors("check bool2double error");
}

void kernel_util::conjunction(bool *b1, bool *b2, int size) {
	conjunction_kernel << <GRID1D(size), BLOCK1D >> > (b1, b2, size);
	kernelUtilLogger.debug("conjunction_kernel invoked");
	cudaCheckErrors("check conjunction error");
}

void kernel_util::disjunction(bool *b1, bool *b2, int size) {
	disjunction_kernel << <GRID1D(size), BLOCK1D >> > (b1, b2, size);
	kernelUtilLogger.debug("disjunction_kernel invoked");
	cudaCheckErrors("check disjunction error");
}

void kernel_util::subtraction(bool *b1, bool *b2, int size) {
	subtraction_kernel << <GRID1D(size), BLOCK1D >> > (b1, b2, size);
	kernelUtilLogger.debug("subtraction_kernel invoked");
	cudaCheckErrors("check subtraction error");
}

void kernel_util::negateConjunctionStar(bool *b1, bool *b2, int size) {
	negateConjunctionStar_kernel << <GRID1D(size), BLOCK1D >> > (b1, b2, size);
	kernelUtilLogger.debug("negate_conjunction_kernel invoked");
	cudaCheckErrors("check negateConjunctionStar error");
}

void kernel_util::ConjunctionStar(bool *b1, bool *b2, int size) {
	conjunctionStar_kernel << <GRID1D(size), BLOCK1D >> > (b1, b2, size);
	kernelUtilLogger.debug("conjunctionStar_kernel invoked");
	cudaCheckErrors("check ConjunctionStar error");
}

void kernel_util::up2down(bool *b1, bool *b2, int size) {
	up2down_kernel << <GRID1D(size), BLOCK1D >> > (b1, b2, size);
	kernelUtilLogger.debug("up2down_kernel invoked");
	cudaCheckErrors("check up2down error");
}

double kernel_util::sum(double *d, int size) {
	sum_kernel << <1, BLOCK1D >> > (d, size);
	double r;
	hipMemcpy(&r, d, sizeof(double), hipMemcpyDeviceToHost);
	kernelUtilLogger.debug("sum_kernel invoked");
	cudaCheckErrors("check sum error");
	return r;
}

void kernel_util::initMaskSignal(bool *b, int initSize, int size) {
	initMaskSignal_kernel << <GRID1D(size), BLOCK1D >> > (b, initSize, size);
	kernelUtilLogger.debug("initMaskSignal_kernel invoked");
	cudaCheckErrors("check initMaskSignal error");
}